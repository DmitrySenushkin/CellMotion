#include "hip/hip_runtime.h"
#include "CellSimulator.cuh"
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <algorithm>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include "kernel_impl.cuh"
#include <hip/hip_runtime.h>

extern "C"
{
	void cudaInit(int argc, char **argv)
	{
		int devID;

		// use command-line specified CUDA device, otherwise use device with highest Gflops/s
		devID = findCudaDevice(argc, (const char **)argv);

		if (devID < 0)
		{
			printf("No CUDA Capable devices found, exiting...\n");
			exit(EXIT_SUCCESS);
		}
	}

	void allocateArray(void **devPtr, size_t size)
	{
		checkCudaErrors(hipMalloc(devPtr, size));
	}

	void freeArray(void *devPtr)
	{
		checkCudaErrors(hipFree(devPtr));
	}

	void threadSync()
	{
		checkCudaErrors(hipDeviceSynchronize());
	}

	void copyArrayToDevice(void *device, const void *host, int offset, int size)
	{
		checkCudaErrors(hipMemcpy((char *)device + offset, host, size, hipMemcpyHostToDevice));
	}

	void copyArrayFromDevice(void *host, const void *device, int size)
	{
		checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
	}

	uint iDivUp(uint a, uint b)
	{
		return (a % b != 0) ? (a / b + 1) : (a / b);
	}

	void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
	{
		numThreads = std::min(blockSize, n);
		numBlocks = iDivUp(n, numThreads);
	}

	void calculateHashValue(uint  *gridParticleHash, uint  *gridParticleIndex, float4 *pos, int numParticles)
	{
		uint numThreads, numBlocks;
		computeGridSize(numParticles, 256, numBlocks, numThreads);

		// execute the kernel
		calculateHashValueDevice << < numBlocks, numThreads >> >(	gridParticleHash,
																	gridParticleIndex,
																	pos,
																	numParticles);
		//hipDeviceSynchronize();
		// check if kernel invocation generated an error
		getLastCudaError("Kernel execution failed: __calculateHashValueDevice__");
	}

	void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
	{
		thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
			thrust::device_ptr<uint>(dGridParticleHash + numParticles),
			thrust::device_ptr<uint>(dGridParticleIndex));

		//hipDeviceSynchronize();
	}


	void integrateSystem(float4 *newPos, float4 * pos, float4 * acc, float deltaTime, uint numParticles, hiprandState *globalState)
	{
		uint numThreads, numBlocks;
		computeGridSize(numParticles, 256, numBlocks, numThreads);

		//execute the kernel
		integrateDevice << < numBlocks, numThreads >> > (newPos, pos, acc, deltaTime, numParticles, globalState);
		
		//hipDeviceSynchronize();

		// check if kernel invocation generated an error
		getLastCudaError("Kernel execution failed: __integrateDevice__ ");
	}

	void setParameters(SimulationParams *hostSimParams, PlaneParams *hostPlaneParams)
	{
		// copy parameters to constant memory
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(simParams), hostSimParams, sizeof(SimulationParams)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(planeParams), hostPlaneParams, sizeof(PlaneParams)));
	}

	void reorderDataAndFindCellStart(uint  *cellStart,
		uint  *cellEnd,
		float4 *sortedPos,
		float4 *sortedAcc,
		uint  *gridParticleHash,
		uint  *gridParticleIndex,
		float4 *oldPos,
		float4 *oldAcc,
		uint   numParticles,
		uint   numCells)
	{
		uint numThreads, numBlocks;
		computeGridSize(numParticles, 256, numBlocks, numThreads);

		// set all cells to empty
		checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));
		checkCudaErrors(hipMemset(cellEnd, 0xffffffff, numCells*sizeof(uint)));

		uint smemSize = sizeof(uint)*(numThreads + 1);
		reorderDataAndFindCellStartDevice << < numBlocks, numThreads, smemSize >> >(
			cellStart,
			cellEnd,
			sortedPos,
			sortedAcc,
			gridParticleHash,
			gridParticleIndex,
			oldPos,
			oldAcc,
			numParticles);

		//hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed: __reorderDataAndFindCellStartDevice_");


	}

	void interact(float4 *newAcc,
		float4 *sortedPos,
		uint  *gridParticleIndex,
		uint  *cellStart,
		uint  *cellEnd,
		uint   numParticles,
		uint   numCells,
		float  damping)
	{

		// thread per particle
		uint numThreads, numBlocks;
		computeGridSize(numParticles, 64, numBlocks, numThreads);

		// execute the kernel
		interactDevice << < numBlocks, numThreads >> >(newAcc,
			sortedPos,
			gridParticleIndex,
			cellStart,
			cellEnd,
			numParticles,
			damping);

		//hipDeviceSynchronize();

		// check if kernel invocation generated an error
		getLastCudaError("Kernel execution failed: __integrateDevice__");
	}

	void initialazeKernelStates(hiprandState * state, unsigned long seed, uint numParticles)
	{
		uint numThreads, numBlocks;
		computeGridSize(numParticles, 256, numBlocks, numThreads);

		initializeCurandOnKernels << < numBlocks, numThreads >> >(state, seed, numParticles);

		getLastCudaError("Kernel execution failed: __initializeCurandOnKernels__");
	}
}